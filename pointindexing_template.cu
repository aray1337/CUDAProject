//nvcc -O2 pointindexing_template.cu -o pointindexing -I /home/microway/cuda-samples/Common/
//./pointindexing  100 2 
//./pointindexing  10000000 10  

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>
#include <thrust/partition.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include <iostream>
#include <iterator>
#include <sys/time.h>
#include <time.h>

typedef unsigned short ushort;
typedef unsigned char uchar;

using namespace std;

float calc_time(char *msg,timeval t0, timeval t1)
{
 	long d = t1.tv_sec*1000000+t1.tv_usec - t0.tv_sec * 1000000-t0.tv_usec;
 	float t=(float)d/1000;
 	if(msg!=NULL)
 		printf("%s ...%10.3f\n",msg,t);
 	return t;
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct point2d
{
    ushort x,y;
    __host__ __device__
    point2d() {}

    __host__ __device__
    point2d(ushort _x, ushort _y) : x(_x), y(_y) {}

};

struct xytor
{

  int lev;

  __host__ __device__
  xytor(int lev): lev(lev) {}

    __host__ __device__
    uint operator()(point2d p )
    {
	    ushort x = p.x;
	    ushort y = p.y;
	    {
		ushort a=x>>(16-lev);
		ushort b=y>>(16-lev);
		return b*(1<<lev)+a;
	    }
    }
};


int main(int argc, char *argv[])
{
    if (argc!=3)
    {
        printf("USAGE: %s #points lev(0<l<16)\n", argv[0]);
        exit(1);
    }
	
    int num_points = atoi(argv[1]);
    if(num_points<0) num_points=10000;    
    printf("num_points=%d\n",num_points);
    
    int run_lev=atoi(argv[2]);
    if(run_lev<0) run_lev=10;    
    printf("run_lev=%d\n",run_lev);
    
    vector<unsigned int> da(num_points);
    timeval s0, s1,s2,s3,s4,s5,s6,s7;
    
    //allocate host memory
    point2d * h_points=new point2d[num_points];
    uint *h_cellids=new uint[num_points];
     
    point2d * dptr_points=NULL;
    uint *dptr_cellids=NULL;
    HANDLE_ERROR( hipMalloc( (void**)&dptr_points,num_points* sizeof(point2d)));
    HANDLE_ERROR( hipMalloc( (void**)&dptr_cellids,num_points* sizeof(uint)));
    assert(dptr_points!=NULL&&dptr_cellids!=NULL);
    
    //generate points
    gettimeofday(&s0, NULL);    
    for(int i=0;i<num_points;i++)
    {
    	point2d p;
    	p.x=random()%65536;
    	p.y=random()%65536;
    	h_points[i]=p;
    }    
    gettimeofday(&s1, NULL);    
    calc_time("generating random points\n",s0,s1);
    
    // copy point data from CPU host to GPU device 
    HANDLE_ERROR( hipMemcpy( dptr_points, h_points, num_points * sizeof(point2d), hipMemcpyHostToDevice ) ); 
    gettimeofday(&s2, NULL);    
    calc_time("trnasferring data to GPU\n",s1,s2);  
        
    thrust::device_ptr<point2d> d_points=thrust::device_pointer_cast(dptr_points);
    thrust::device_ptr<uint> d_cellids =thrust::device_pointer_cast(dptr_cellids);
    
    //====================================================================================================
    //YOUR WORK below: Step 1- transform point coordinates to cell identifiers; pay attention to functor xytor
    //thrust::transform(...);
    int keys[num_points] = {};
    xytor xytor_init(num_points); // num__points just placeholder....
    thrust::transform(h_points, h_points + num_points, keys, xytor_init);
    hipDeviceSynchronize();
    gettimeofday(&s3, NULL);    
    calc_time("transforming..............\n",s2,s3);

    
    //YOUR WORK below: Step 2- sort (cellid,point) pairs 
    //thrust::stable_sort_by_key(...)
    // thrust::stable_sort_by_key(thrust::host, keys, keys + num_points, h_points, thrust::greater<int>());
    hipDeviceSynchronize();
    gettimeofday(&s4, NULL);    
    calc_time("sorting..............\n",s3,s4);
    
    uint *dptr_PKey=NULL;
    uint *dptr_PLen=NULL;
    uint *dptr_PPos=NULL;    
    HANDLE_ERROR( hipMalloc( (void**)&dptr_PKey,num_points* sizeof(uint)));
    HANDLE_ERROR( hipMalloc( (void**)&dptr_PLen,num_points* sizeof(uint)));
    HANDLE_ERROR( hipMalloc( (void**)&dptr_PPos,num_points* sizeof(uint)));
    assert(dptr_PKey!=NULL&&dptr_PLen!=NULL&&dptr_PPos!=NULL);
    thrust::device_ptr<uint> d_PKey =thrust::device_pointer_cast(dptr_PKey);
    thrust::device_ptr<uint> d_PLen=thrust::device_pointer_cast(dptr_PLen);
    thrust::device_ptr<uint> d_PPos=thrust::device_pointer_cast(dptr_PPos);
    
    //YOUR WORK below: Step 3- reduce by key 
    //use  d_cellids as the first input vector and thrust::constant_iterator<int>(1) as the second input
    size_t num_cells=0;//num_cells is initialized to 0 just to make the template compile; it should be updated next
    // num_cells = thrust::reduce_by_key(...).first - d_PKey
    // num_cells = thrust::reduce_by_key(keys, keys - d_PKey, num_points, binary_pred );	
    hipDeviceSynchronize();
    gettimeofday(&s5, NULL);
    calc_time("reducing.......\n",s4,s5);
    
    //YOUR WORK below: Step 4-  exclusive scan using d_PLen as the input and d_PPos as the output
    //thrust::exclusive_scan(...)
    // thrust::exclusive_scan(d_PLen, d_PLen + num_points, d_PPos);
    hipDeviceSynchronize();
    gettimeofday(&s6, NULL);
    calc_time("scan.......\n",s5,s6); 
    //====================================================================================================
    //transferring data back to CPU
    uint *h_PKey=new uint[num_cells];
    uint *h_PLen=new uint[num_cells];
    uint *h_PPos=new uint[num_cells];    
    HANDLE_ERROR( hipMemcpy( h_points, dptr_points, num_points * sizeof(point2d), hipMemcpyDeviceToHost) ); 
    HANDLE_ERROR( hipMemcpy( h_cellids, dptr_cellids, num_points * sizeof(uint), hipMemcpyDeviceToHost) ); 
    HANDLE_ERROR( hipMemcpy( h_PKey, dptr_PKey, num_cells * sizeof(uint), hipMemcpyDeviceToHost) ); 
    HANDLE_ERROR( hipMemcpy( h_PLen, dptr_PLen, num_cells * sizeof(uint), hipMemcpyDeviceToHost) ); 
    HANDLE_ERROR( hipMemcpy( h_PPos, dptr_PPos, num_cells * sizeof(uint), hipMemcpyDeviceToHost) ); 
    gettimeofday(&s7, NULL);
    calc_time("transferring back to CPU.......\n",s6,s7);     
    
    //you would have to override the output opertor of point2d to output points to std::cout
    //thrust::copy(h_points, h_points+num_cells, std::ostream_iterator<point2d>(std::cout, " "));    
    
    //alternatively, you can access h_points array and print out x/y
    int point_out=(num_points>50)?50:num_points;
    for(int i=0;i<point_out;i++)
    {
     	point2d p=h_points[i];
     	printf("(%d,%d)",p.x,p.y);
     }
    printf("\n");
     
    cout<<"cell identifiers:";
    thrust::copy(h_cellids, h_cellids+point_out, std::ostream_iterator<uint>(std::cout, " "));       
    cout<<endl;
    int cell_out=(num_cells>20)?20:num_cells;
    cout<<"unique cell identifiers:";
    thrust::copy(h_PKey, h_PKey+cell_out, std::ostream_iterator<uint>(std::cout, " "));       
    cout<<endl;
    cout<<"number of points in cells:";
    thrust::copy(h_PLen, h_PLen+cell_out, std::ostream_iterator<uint>(std::cout, " "));       
    cout<<endl;
    cout<<"starting point position in cells:";
    thrust::copy(h_PPos, h_PPos+cell_out, std::ostream_iterator<uint>(std::cout, " "));       
    cout<<endl;
     
     //clean up
    hipFree(dptr_points);
    hipFree(dptr_cellids);
    hipFree(dptr_PKey);
    hipFree(dptr_PLen);
    hipFree(dptr_PPos);
    delete[] h_points;
    delete[] h_cellids;
    delete[] h_PKey;
    delete[] h_PLen;
    delete[] h_PPos;
}